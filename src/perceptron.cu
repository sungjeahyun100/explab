#include "hip/hip_runtime.h"
#include "perceptron.hpp"

// 현재 시간 문자열 반환 (가중치 저장 파일명에 사용)
std::string getCurrentTimestamp()
{
    auto now = std::chrono::system_clock::now();
    std::time_t t_now = std::chrono::system_clock::to_time_t(now);
    std::tm tm_now = *std::localtime(&t_now);

    std::ostringstream oss;
    oss << std::put_time(&tm_now, "%Y-%m-%d_%H%M%S");
    return oss.str();
}

// 가중치 파일에서 weight, bias 불러오기
void perceptronLayer::loadWeight(const std::string &path)
{
    std::ifstream test_subject(path, std::ios::binary);
    if (!test_subject) {
        std::cerr << "Error opening file: " << path << std::endl;
        return;
    }

    test_subject >> weight;
    test_subject >> bias;

    test_subject.close();
}

// 가중치 파일로 저장 (이름: subject+타임스탬프)
void perceptronLayer::saveWeight()
{
    std::ofstream test_subject(WEIGHT_DATAPATH + "subject" + getCurrentTimestamp() + ".bin");
    test_subject << weight;
    test_subject << bias;
    test_subject.close();
}

d_matrix<double>& perceptronLayer::getOutput() { return output; }

// weight, bias를 GPU로 복사
void perceptronLayer::updateWeightInDev() {
    weight.cpyToDev();
    bias.cpyToDev();
}

// feedforward: z = W x + b, output = z
// (활성화는 ActivateLayer에서 적용)
void perceptronLayer::feedforward(const d_matrix<double>& raw_input) {
    input = raw_input;

    z = matrixPlus(matrixMP(weight, input), bias);

    output = z;
}

// 그래디언트 계산 (델타, Gt_W, Gt_B)
// 델타: δ = (next->weight^T * next->delta) ⊙ act_deriv
// Gt_W = δ * input^T, Gt_B = δ
void perceptronLayer::calculateGrad(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv) {
    d_matrix<double> grad_input = external_delta;

    if (next != nullptr) {
        d_matrix<double> weighted_delta = matrixMP(next->weight.transpose(), next->delta);
        weighted_delta.cpyToDev();
        grad_input = weighted_delta;
    }

    delta = HadamardProduct(grad_input, act_deriv);
    delta.cpyToDev();

    Gt_W = matrixMP<double>(delta, input.transpose());
    Gt_B = delta;
    Gt_W.cpyToDev();
    Gt_B.cpyToDev();

    hipDeviceSynchronize();
}

// 입력 설정 (input = in)
void ActivateLayer::pushInput(const d_matrix<double>& in){
    input = in;
    input.cpyToDev();
}

// 활성화 적용 (output = f(input))
// 지원: ReLU, LReLU, Identity, Sigmoid
void ActivateLayer::Active(){
    switch (act) {
        case ActivationType::ReLU:
            output = MatrixActivate<double, relu>(input); break;
        case ActivationType::LReLU:
            output = MatrixActivate<double, lrelu>(input); break;
        case ActivationType::Identity:
            output = MatrixActivate<double, Identity>(input); break;
        case ActivationType::Sigmoid:
            output = MatrixActivate<double, sigmoid>(input); break;
        case ActivationType::Tanh:
            output = MatrixActivate<double, Tanh>(input); break;
        default:
            throw std::runtime_error("Unsupported ActivationType in perceptronLayer");
    }
}

// 활성화 함수 미분값 반환 (f'(z))
// ReLU: 1(x>0), 0(x<=0)
// LReLU: 1(x>0), 0.01(x<=0)
// Identity: 1
// Sigmoid: σ'(x) = σ(x)(1-σ(x))
d_matrix<double> ActivateLayer::d_Active(const d_matrix<double>& z) {
    switch (act) {
        case ActivationType::ReLU:
            return MatrixActivate<double, d_relu>(z);
        case ActivationType::LReLU:
            return MatrixActivate<double, d_lrelu>(z);
        case ActivationType::Identity:
            return MatrixActivate<double, d_I>(z);
        case ActivationType::Sigmoid:
            return MatrixActivate<double, d_sigmoid>(z);
        case ActivationType::Tanh:
            return MatrixActivate<double, d_tanh>(z);
        default:
            throw std::runtime_error("Unsupported ActivationType in d_Active");
    }
}

// 활성화 결과 반환
const d_matrix<double>& ActivateLayer::getOutput() const {
    return output; 
}

// 타겟 입력
void LossLayer::pushTarget(const d_matrix<double>& Target){
    target = Target;
}

// 출력 입력
void LossLayer::pushOutput(const d_matrix<double>& Output){
    output = Output;
}

// 손실값 반환
// MSE: L = 1/n Σ(y-p)^2
// CrossEntropy: L = -Σ y log(softmax(p))
double LossLayer::getLoss(){
    switch (Loss)
    {
    case LossType::MSE: {
        d_matrix<double> diff = matrixPlus(output, ScalaProduct(target, -1.0));
        diff.cpyToDev();
        d_matrix<double> squared = HadamardProduct(diff, diff);
        squared.cpyToDev();
        return plusAllElements(squared) / static_cast<double>(output.getRow());
    }

    case LossType::CrossEntropy: {
        d_matrix<double> prob = softmax(output);
        prob.cpyToHost();
        target.cpyToHost();

        double loss = 0.0;
        for (int i = 0; i < target.getRow(); ++i) {
            double y = target(i, 0);
            double p = prob(i, 0);
            if (y == 1.0) {
                loss -= std::log(std::max(p, 1e-15));
            }
        }
        return loss;
    }

    default:
        throw std::runtime_error("Unsupported LossType in calculateLoss");
    }
}

// 손실 미분 반환
// MSE: dL/dz = 2(y-p)
// CrossEntropy: dL/dz = softmax(p) - y
d_matrix<double> LossLayer::getGrad() {
    switch (Loss){
    case LossType::MSE: {
        d_matrix<double> diff = matrixPlus(output, ScalaProduct(target, -1.0));
        return ScalaProduct(diff, 2.0);
    }

    case LossType::CrossEntropy: {
        d_matrix<double> prob = softmax(output);
        return matrixPlus(prob, ScalaProduct(target, -1.0));
    }

    default:
        throw std::runtime_error("Unsupported LossType in getGrad");
    }
}

Adam::~Adam(){}

// Adam 옵티마이저 역전파
// m, v: 1차/2차 모멘트, 베타1/2, epsilon, t(스텝)
// 업데이트 수식:
// m = β₁ m + (1-β₁)g, v = β₂ v + (1-β₂)g²
// m̂ = m/(1-β₁ᵗ), v̂ = v/(1-β₂ᵗ)
// W -= lr * m̂/(sqrt(v̂)+ε)
void Adam::backprop(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv)
{
    this->calculateGrad(next, external_delta, act_deriv);
    m_W = matrixPlus(ScalaProduct(m_W, beta1), ScalaProduct(this->Gt_W, 1.0 - beta1));
    v_W = matrixPlus(ScalaProduct(v_W, beta2), ScalaProduct(HadamardProduct(this->Gt_W, this->Gt_W), 1.0 - beta2));
    m_B = matrixPlus(ScalaProduct(m_B, beta1), ScalaProduct(this->Gt_B, 1.0 - beta1));
    v_B = matrixPlus(ScalaProduct(v_B, beta2), ScalaProduct(HadamardProduct(this->Gt_B, this->Gt_B), 1.0 - beta2));
    double beta1t = 1.0 - std::pow(beta1, t);
    double beta2t = 1.0 - std::pow(beta2, t);
    d_matrix<double> m_W_hat = ScalaProduct(m_W, 1.0 / beta1t);
    d_matrix<double> v_W_hat = ScalaProduct(v_W, 1.0 / beta2t);
    d_matrix<double> m_B_hat = ScalaProduct(m_B, 1.0 / beta1t);
    d_matrix<double> v_B_hat = ScalaProduct(v_B, 1.0 / beta2t);
    m_W_hat.cpyToDev();
    v_W_hat.cpyToDev();
    m_B_hat.cpyToDev();
    v_B_hat.cpyToDev();
    this->weight = matrixPlus(
        this->weight,
        ScalaProduct(HadamardProduct(m_W_hat, MatrixActivate<double, devide>(ScalaPlus(MatrixActivate<double, sqr>(v_W_hat), epsilon))), (-1)*this->learning_rate)
    );
    this->bias = matrixPlus(
        this->bias,
        ScalaProduct(HadamardProduct(m_B_hat, MatrixActivate<double, devide>(ScalaPlus(MatrixActivate<double, sqr>(v_B_hat), epsilon))), (-1)*this->learning_rate)
    );
    this->updateWeightInDev();
    hipDeviceSynchronize();
    t++;
}

SGD::~SGD(){}

// SGD 옵티마이저 역전파
// W -= lr * grad
void SGD::backprop(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv)
{
    this->calculateGrad(next, external_delta, act_deriv);
    this->weight = matrixPlus(this->weight, ScalaProduct(this->Gt_W, (-1) * this->learning_rate));
    this->bias = matrixPlus(this->bias, ScalaProduct(this->Gt_B, (-1) * this->learning_rate));
    this->updateWeightInDev();
    hipDeviceSynchronize();
}

/*
[MLP(다층 퍼셉트론) 구성 예시]

// 1. 계층 선언 (입력, 은닉, 출력)
SGD input_layer(입크기, 은닉크기, lr, InitType::He);
ActivateLayer act1(은닉크기, 1, ActivationType::ReLU);
SGD output_layer(은닉크기, 출력크기, lr, InitType::He);
ActivateLayer act2(출력크기, 1, ActivationType::Sigmoid); // 또는 Softmax
LossLayer loss(출력크기, 1, LossType::CrossEntropy);

// 2. 순전파 예시
input_layer.feedforward(input); // 첫 계층
act1.pushInput(input_layer.getOutput());
act1.Active();
output_layer.feedforward(act1.getOutput());
act2.pushInput(output_layer.getOutput());
act2.Active();

// 3. 역전파 예시
loss.pushTarget(target);
loss.pushOutput(act2.getOutput());
d_matrix<double> grad = loss.getGrad();
output_layer.backprop(nullptr, grad, act2.d_Active(output_layer.getOutput()));
input_layer.backprop(&output_layer, output_layer.delta, act1.d_Active(input_layer.getOutput()));

[MLP(다층 퍼셉트론) 구성 예시 - Adam 사용]

// 1. 계층 선언 (입력, 은닉, 출력)
Adam input_layer(입크기, 은닉크기, lr, InitType::He);
ActivateLayer act1(은닉크기, 1, ActivationType::ReLU);
Adam output_layer(은닉크기, 출력크기, lr, InitType::He);
ActivateLayer act2(출력크기, 1, ActivationType::Sigmoid); // 또는 Softmax
LossLayer loss(출력크기, 1, LossType::CrossEntropy);

// 2. 순전파 예시
input_layer.feedforward(input); // 첫 계층
act1.pushInput(input_layer.getOutput());
act1.Active();
output_layer.feedforward(act1.getOutput());
act2.pushInput(output_layer.getOutput());
act2.Active();

// 3. 역전파 예시
loss.pushTarget(target);
loss.pushOutput(act2.getOutput());
d_matrix<double> grad = loss.getGrad();
output_layer.backprop(nullptr, grad, act2.d_Active(output_layer.getOutput()));
input_layer.backprop(&output_layer, output_layer.delta, act1.d_Active(input_layer.getOutput()));
*/



