#include "hip/hip_runtime.h"
#include "perceptron.hpp"

// 현재 시간 문자열 반환 (가중치 저장 파일명에 사용)
std::string getCurrentTimestamp()
{
    auto now = std::chrono::system_clock::now();
    std::time_t t_now = std::chrono::system_clock::to_time_t(now);
    std::tm tm_now = *std::localtime(&t_now);

    std::ostringstream oss;
    oss << std::put_time(&tm_now, "%Y-%m-%d_%H%M%S");
    return oss.str();
}

// 가중치 파일에서 weight, bias 불러오기
void perceptronLayer::loadWeight(const std::string &path)
{
    std::ifstream test_subject(path, std::ios::binary);
    if (!test_subject) {
        std::cerr << "Error opening file: " << path << std::endl;
        return;
    }

    test_subject >> weight;
    test_subject >> bias;

    test_subject.close();
}

// 가중치 파일로 저장 (이름: subject+타임스탬프)
void perceptronLayer::saveWeight()
{
    std::ofstream test_subject(WEIGHT_DATAPATH + "subject" + getCurrentTimestamp() + ".bin");
    test_subject << weight;
    test_subject << bias;
    test_subject.close();
}

d_matrix<double>& perceptronLayer::getOutput() { return output; }

// weight, bias를 GPU로 복사
void perceptronLayer::updateWeightInDev() {
    weight.cpyToDev();
    bias.cpyToDev();
}

// feedforward: z = W x + b, output = z
// (활성화는 ActivateLayer에서 적용)
void perceptronLayer::feedforward(const d_matrix<double>& raw_input) {
    input = raw_input;

    z = matrixPlus(matrixMP(weight, input), bias);

    output = z;
}

// 그래디언트 계산 (델타, Gt_W, Gt_B)
// 델타: δ = (next->weight^T * next->delta) ⊙ act_deriv
// Gt_W = δ * input^T, Gt_B = δ
void perceptronLayer::calculateGrad(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv) {
    d_matrix<double> grad_input = external_delta;

    if (next != nullptr) {
        d_matrix<double> weighted_delta = matrixMP(next->weight.transpose(), next->delta);
        weighted_delta.cpyToDev();
        grad_input = weighted_delta;
    }

    delta = HadamardProduct(grad_input, act_deriv);
    delta.cpyToDev();

    Gt_W = matrixMP<double>(delta, input.transpose());
    Gt_B = delta;

    hipDeviceSynchronize();
}

// 입력 설정 (input = in)
void ActivateLayer::pushInput(const d_matrix<double>& in){
    input = in;
    input.cpyToDev();
}

// 활성화 적용 (output = f(input))
// 지원: ReLU, LReLU, Identity, Sigmoid
void ActivateLayer::Active(){
    switch (act) {
        case ActivationType::ReLU:
            output = MatrixActivate<double, relu>(input); break;
        case ActivationType::LReLU:
            output = MatrixActivate<double, lrelu>(input); break;
        case ActivationType::Identity:
            output = MatrixActivate<double, Identity>(input); break;
        case ActivationType::Sigmoid:
            output = MatrixActivate<double, sigmoid>(input); break;
        case ActivationType::Tanh:
            output = MatrixActivate<double, Tanh>(input); break;
        default:
            throw std::runtime_error("Unsupported ActivationType in perceptronLayer");
    }
}

// 활성화 함수 미분값 반환 (f'(z))
// ReLU: 1(x>0), 0(x<=0)
// LReLU: 1(x>0), 0.01(x<=0)
// Identity: 1
// Sigmoid: σ'(x) = σ(x)(1-σ(x))
d_matrix<double> ActivateLayer::d_Active(const d_matrix<double>& z) {
    switch (act) {
        case ActivationType::ReLU:
            return MatrixActivate<double, d_relu>(z);
        case ActivationType::LReLU:
            return MatrixActivate<double, d_lrelu>(z);
        case ActivationType::Identity:
            return MatrixActivate<double, d_I>(z);
        case ActivationType::Sigmoid:
            return MatrixActivate<double, d_sigmoid>(z);
        case ActivationType::Tanh:
            return MatrixActivate<double, d_tanh>(z);
        default:
            throw std::runtime_error("Unsupported ActivationType in d_Active");
    }
}

// 활성화 결과 반환
const d_matrix<double>& ActivateLayer::getOutput() const {
    return output; 
}

// 타겟 입력
void LossLayer::pushTarget(const d_matrix<double>& Target){
    target = Target;
}

// 출력 입력
void LossLayer::pushOutput(const d_matrix<double>& Output){
    output = Output;
}

// 손실값 반환
// MSE: L = 1/n Σ(y-p)^2
// CrossEntropy: L = -Σ y log(softmax(p))
double LossLayer::getLoss(){
    // 1) 디바이스→호스트 복사
    output.cpyToHost();
    target.cpyToHost();

    switch (Loss)
    {
        case LossType::MSE: {
            // MSE: L = 1/N Σ (output − target)², 전부 호스트 계산
            int N = output.getRow();
            double sum = 0.0;
            for (int i = 0; i < N; ++i) {
                double diff = output(i, 0) - target(i, 0);
                sum += diff * diff;
            }
            return sum / static_cast<double>(N);
        }

        case LossType::CrossEntropy: {
            // 이 구현은 “이진 크로스엔트로피” (비트 단위 분류) 예시입니다.
            // 필요하다면 multi-class softmax 버전으로 바꾸시면 됩니다.
            int N = output.getRow();
            double loss = 0.0;
            for (int i = 0; i < N; ++i) {
                double z = output(i, 0);
                double y = target(i, 0);
                // sigmoid로 확률 p 계산
                double p = 1.0 / (1.0 + std::exp(-z));
                // log(0) 방지용 클리핑
                p = std::min(std::max(p, 1e-7), 1.0 - 1e-7);
                loss += -(y * std::log(p) + (1.0 - y) * std::log(1.0 - p));
            }
            return loss;
        }

        default:
            throw std::runtime_error("Unsupported LossType in getLoss");
    }
}

// 손실 미분 반환
// MSE: dL/dz = 2(y-p)
// CrossEntropy: dL/dz = softmax(p) - y
d_matrix<double> LossLayer::getGrad() {
    // 1) 디바이스→호스트 복사
    output.cpyToHost();
    target.cpyToHost();

    switch (Loss) {
        case LossType::MSE: {
            // L = (1/N) Σ (o - t)^2  이므로  dL/dz = 2*(o - t)/N
            int N = output.getRow();
            // diff = output - target
            d_matrix<double> diff = matrixPlus(output, ScalaProduct(target, -1.0));
            return ScalaProduct(diff, 2.0 / static_cast<double>(N));
        }

        case LossType::CrossEntropy: {
            // 이진 크로스엔트로피 (BCE) + Sigmoid 로 구현
            int N = output.getRow();
            d_matrix<double> grad(N, 1);
            for (int i = 0; i < N; ++i) {
                double z = output(i, 0);
                double y = target(i, 0);
                // Sigmoid 확률
                double p = 1.0 / (1.0 + std::exp(-z));
                // gradient = p - y
                grad(i, 0) = p - y;
            }
            return grad;
        }

        default:
            throw std::runtime_error("Unsupported LossType in getGrad");
    }
}
Adam::~Adam(){}

// Adam 옵티마이저 역전파
// m, v: 1차/2차 모멘트, 베타1/2, epsilon, t(스텝)
// 업데이트 수식:
// m = β₁ m + (1-β₁)g, v = β₂ v + (1-β₂)g²
// m̂ = m/(1-β₁ᵗ), v̂ = v/(1-β₂ᵗ)
// W -= lr * m̂/(sqrt(v̂)+ε)
void Adam::backprop(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv){

    this->t++;

    // 1) gradient 계산
    this->calculateGrad(next, external_delta, act_deriv);

    // 2) 1차 및 2차 모멘트 갱신
    this->m_W = matrixPlus(ScalaProduct(this->m_W, this->beta1), ScalaProduct(this->Gt_W, 1.0 - this->beta1));
    this->v_W = matrixPlus(ScalaProduct(this->v_W, this->beta2), ScalaProduct(HadamardProduct(this->Gt_W, this->Gt_W), 1.0 - this->beta2));
    this->m_B = matrixPlus(ScalaProduct(this->m_B, this->beta1), ScalaProduct(this->Gt_B, 1.0 - this->beta1));
    this->v_B = matrixPlus(ScalaProduct(this->v_B, this->beta2), ScalaProduct(HadamardProduct(this->Gt_B, this->Gt_B), 1.0 - this->beta2));

    // 3) 편향 보정 계수
    double bias_corr1 = 1.0 - std::pow(this->beta1, this->t);
    double bias_corr2 = 1.0 - std::pow(this->beta2, this->t);

    // 4) 편향 보정된 모멘트
    d_matrix<double> m_W_hat = ScalaProduct(this->m_W, 1.0 / bias_corr1);
    d_matrix<double> v_W_hat = ScalaProduct(this->v_W, 1.0 / bias_corr2);
    d_matrix<double> m_B_hat = ScalaProduct(this->m_B, 1.0 / bias_corr1);
    d_matrix<double> v_B_hat = ScalaProduct(this->v_B, 1.0 / bias_corr2);

    // 5) 분모: sqrt(v̂) + ε
    //    MatrixActivate<sqrt> 는 elementwise sqrt, devide 는 reciprocal
    auto sqrt_vW = MatrixActivate<double, sqr>(v_W_hat);
    auto denomW  = ScalaPlus(sqrt_vW, this->epsilon);
    auto invDenW = MatrixActivate<double, devide>(denomW);

    auto sqrt_vB = MatrixActivate<double, sqr>(v_B_hat);
    auto denomB  = ScalaPlus(sqrt_vB, this->epsilon);
    auto invDenB = MatrixActivate<double, devide>(denomB);

    // 6) 파라미터 업데이트
    //    w ← w − lr * (m̂ ⊙ invDen)
    this->weight = matrixPlus(
        this->weight,
        ScalaProduct(HadamardProduct(m_W_hat, invDenW), -this->learning_rate)
    );
    this->bias = matrixPlus(
        this->bias,
        ScalaProduct(HadamardProduct(m_B_hat, invDenB), -this->learning_rate)
    );

    // 7) 디바이스 메모리에 복사
    this->updateWeightInDev();
    hipDeviceSynchronize();

}

SGD::~SGD(){}

// SGD 옵티마이저 역전파
// W -= lr * grad
void SGD::backprop(perceptronLayer* next, const d_matrix<double>& external_delta, const d_matrix<double>& act_deriv)
{
    this->calculateGrad(next, external_delta, act_deriv);
    this->weight = matrixPlus(this->weight, ScalaProduct(this->Gt_W, (-1) * this->learning_rate));
    this->bias = matrixPlus(this->bias, ScalaProduct(this->Gt_B, (-1) * this->learning_rate));
    this->updateWeightInDev();
    hipDeviceSynchronize();
}

/*
[MLP(다층 퍼셉트론) 구성 예시]

// 1. 계층 선언 (입력, 은닉, 출력)
SGD input_layer(입크기, 은닉크기, lr, InitType::He);
ActivateLayer act1(은닉크기, 1, ActivationType::ReLU);
SGD output_layer(은닉크기, 출력크기, lr, InitType::He);
ActivateLayer act2(출력크기, 1, ActivationType::Sigmoid); // 또는 Softmax
LossLayer loss(출력크기, 1, LossType::CrossEntropy);

// 2. 순전파 예시
input_layer.feedforward(input); // 첫 계층
act1.pushInput(input_layer.getOutput());
act1.Active();
output_layer.feedforward(act1.getOutput());
act2.pushInput(output_layer.getOutput());
act2.Active();

// 3. 역전파 예시
loss.pushTarget(target);
loss.pushOutput(act2.getOutput());
d_matrix<double> grad = loss.getGrad();
output_layer.backprop(nullptr, grad, act2.d_Active(output_layer.getOutput()));
input_layer.backprop(&output_layer, output_layer.delta, act1.d_Active(input_layer.getOutput()));

[MLP(다층 퍼셉트론) 구성 예시 - Adam 사용]

// 1. 계층 선언 (입력, 은닉, 출력)
Adam input_layer(입크기, 은닉크기, lr, InitType::He);
ActivateLayer act1(은닉크기, 1, ActivationType::ReLU);
Adam output_layer(은닉크기, 출력크기, lr, InitType::He);
ActivateLayer act2(출력크기, 1, ActivationType::Sigmoid); // 또는 Softmax
LossLayer loss(출력크기, 1, LossType::CrossEntropy);

// 2. 순전파 예시
input_layer.feedforward(input); // 첫 계층
act1.pushInput(input_layer.getOutput());
act1.Active();
output_layer.feedforward(act1.getOutput());
act2.pushInput(output_layer.getOutput());
act2.Active();

// 3. 역전파 예시
loss.pushTarget(target);
loss.pushOutput(act2.getOutput());
d_matrix<double> grad = loss.getGrad();
output_layer.backprop(nullptr, grad, act2.d_Active(output_layer.getOutput()));
input_layer.backprop(&output_layer, output_layer.delta, act1.d_Active(input_layer.getOutput()));
*/



