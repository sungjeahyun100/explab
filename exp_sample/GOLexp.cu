#include "hip/hip_runtime.h"
#include <database.hpp>
#include <perceptron.hpp>

const std::string path = "../dataset/result";

// 데이터셋 로드 함수: dataset/sampleN.txt 파일에서
// 10x10 입력 패턴과 300x300 결과 패턴을 읽어온다.
std::vector<std::pair<d_matrix<double>, d_matrix<double>>> loadPatternData(int count){
    std::vector<std::pair<d_matrix<double>, d_matrix<double>>> data;
    data.reserve(count);

    for(int idx = 1; idx <= count; ++idx){
        std::string path = "../dataset/sample" + std::to_string(idx) + ".txt";
        std::ifstream fin(path);
        if(!fin){
            std::cerr << "파일을 열 수 없습니다: " << path << std::endl;
            continue;
        }

        d_matrix<double> inputMat(100,1);      // 10x10 패턴
        d_matrix<double> outputMat(BOARDHEIGHT*BOARDWIDTH,1); // 300x300 결과 패턴

        std::string line;
        // 입력 패턴 읽기
        for(int r=0;r<10 && std::getline(fin,line); ++r){
            for(int c=0;c<10 && c<(int)line.size(); ++c){
                inputMat(r*10+c,0) = line[c]-'0';
            }
        }

        // 라벨(생존 칸 수)은 현재 사용하지 않으므로 읽기만 하고 무시
        std::getline(fin,line); // label
        std::getline(fin,line); // blank line

        // 결과 패턴 읽기 (300줄)
        for(int r=0;r<BOARDHEIGHT && std::getline(fin,line); ++r){
            for(int c=0;c<BOARDWIDTH && c<(int)line.size(); ++c){
                outputMat(r*BOARDWIDTH+c,0) = line[c]-'0';
            }
        }

        data.emplace_back(inputMat, outputMat);
    }

    return data;
}

int main(){
    Adam inputlayer(100, 256, 0.011, InitType::He);
    ActivateLayer input(256, 1, ActivationType::LReLU);
    Adam hiddenlayer1(256, 512, 0.011, InitType::He);
    ActivateLayer hidden1(512, 1, ActivationType::LReLU);
    Adam outputlayer(512, BOARDHEIGHT*BOARDWIDTH, 0.011, InitType::He);
    ActivateLayer output(BOARDHEIGHT*BOARDWIDTH, 1, ActivationType::LReLU);
    LossLayer loss(BOARDHEIGHT*BOARDWIDTH, 1, LossType::MSE);

    // 데이터셋 로드 (sample1 ~ sample40)
    auto dataset = loadPatternData(40);

    std::filesystem::create_directories(path);

    const char *commend1 = "find ../dataset/result -type f -delete";

    std::system(commend1);

    const int epochs = 5;           // 에폭 수 (예시)
    const int batchSize = 10;       // 미니배치 크기

    std::mt19937 rng(std::random_device{}());

    // 학습 루프
    for(int epoch=0; epoch<epochs; ++epoch){
        auto startTime = std::chrono::steady_clock::now();
        std::shuffle(dataset.begin(), dataset.end(), rng);

        for(size_t i=0; i<dataset.size(); i += batchSize){
            size_t end = std::min(i+batchSize, dataset.size());
            for(size_t j=i; j<end; ++j){
                auto& inputMat = dataset[j].first;
                auto& targetMat = dataset[j].second;

                // 순전파
                inputlayer.feedforward(inputMat);
                input.pushInput(inputlayer.getOutput());
                input.Active();

                hiddenlayer1.feedforward(input.getOutput());
                hidden1.pushInput(hiddenlayer1.getOutput());
                hidden1.Active();

                outputlayer.feedforward(hidden1.getOutput());
                output.pushInput(outputlayer.getOutput());
                output.Active();

                // 손실 계산 및 역전파
                loss.pushTarget(targetMat);
                loss.pushOutput(output.getOutput());
                d_matrix<double> grad = loss.getGrad();

                outputlayer.backprop(nullptr, grad, output.d_Active(outputlayer.getOutput()));
                // 다음 계층 정보를 포인터로 전달하면 내부에서 next->delta를 활용하여
                // 역전파가 진행된다. 외부에서 delta에 접근할 필요는 없다.
                d_matrix<double> dummy(1,1); // 사용되지 않음
                hiddenlayer1.backprop(&outputlayer, dummy, hidden1.d_Active(hiddenlayer1.getOutput()));
                inputlayer.backprop(&hiddenlayer1, dummy, input.d_Active(inputlayer.getOutput()));

                printProgressBar(j, dataset.size(), startTime, "Epoch" + std::to_string(epoch+1) + "진행중...");
            }
        }
        std::cout << "✅ Epoch " << (epoch+1)
                  << " 완료! (소요 "
                  << std::chrono::duration_cast<std::chrono::seconds>(
                         std::chrono::steady_clock::now() - startTime
                     ).count()
                  << "초)                                                                                     "
                  << std::endl;
    }

    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto& inputMat = dataset[idx].first;

        inputlayer.feedforward(inputMat);
        input.pushInput(inputlayer.getOutput());
        input.Active();
        hiddenlayer1.feedforward(input.getOutput());
        hidden1.pushInput(hiddenlayer1.getOutput());
        hidden1.Active();
        outputlayer.feedforward(hidden1.getOutput());
        output.pushInput(outputlayer.getOutput());
        output.Active();

        d_matrix<double> pred = output.getOutput();
        pred.cpyToHost();

        std::ofstream result(path + "/semple_result" + std::to_string(idx+1) + ".txt");

        result << "=== sample " << idx+1 << " 결과 ===" << std::endl;
        for(int r=0;r<BOARDHEIGHT;r++){
            for(int c=0;c<BOARDWIDTH;c++){
                result << (pred(r*BOARDWIDTH+c,0) > 0.5 ? '1' : '0');
            }
            result << '\n';
        }
        result << '\n';
        result.close();
    }

    return 0;
}