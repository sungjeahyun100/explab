#include "hip/hip_runtime.h"
#include "database.hpp"
#include "perceptron.hpp"

Adam inputlayer(100, 256, 0.001, InitType::He);
ActivateLayer input(256, 1, ActivationType::LReLU);
Adam hiddenlayer1(256, 256, 0.001, InitType::He);
ActivateLayer hidden1(256, 1, ActivationType::LReLU);
Adam outputlayer(256, BOARDHEIGHT*BOARDWIDTH, 0.001, InitType::He);
ActivateLayer output(BOARDHEIGHT*BOARDWIDTH, 1, ActivationType::LReLU);
LossLayer loss(BOARDHEIGHT*BOARDWIDTH, 1, LossType::MSE);



// 데이터셋 로드 함수: dataset/sampleN.txt 파일에서
// 10x10 입력 패턴과 300x300 결과 패턴을 읽어온다.
std::vector<std::pair<d_matrix<double>, d_matrix<double>>> loadPatternData(int count){
    std::vector<std::pair<d_matrix<double>, d_matrix<double>>> data;
    data.reserve(count);

    for(int idx = 1; idx <= count; ++idx){
        std::string path = "dataset/sample" + std::to_string(idx) + ".txt";
        std::ifstream fin(path);
        if(!fin){
            std::cerr << "파일을 열 수 없습니다: " << path << std::endl;
            continue;
        }

        d_matrix<double> inputMat(100,1);      // 10x10 패턴
        d_matrix<double> outputMat(BOARDHEIGHT*BOARDWIDTH,1); // 300x300 결과 패턴

        std::string line;
        // 입력 패턴 읽기
        for(int r=0;r<10 && std::getline(fin,line); ++r){
            for(int c=0;c<10 && c<(int)line.size(); ++c){
                inputMat(r*10+c,0) = line[c]-'0';
            }
        }

        // 라벨(생존 칸 수)은 현재 사용하지 않으므로 읽기만 하고 무시
        std::getline(fin,line); // label
        std::getline(fin,line); // blank line

        // 결과 패턴 읽기 (300줄)
        for(int r=0;r<BOARDHEIGHT && std::getline(fin,line); ++r){
            for(int c=0;c<BOARDWIDTH && c<(int)line.size(); ++c){
                outputMat(r*BOARDWIDTH+c,0) = line[c]-'0';
            }
        }

        data.emplace_back(inputMat, outputMat);
    }

    return data;
}

int main(){
    // 데이터셋 로드 (sample1 ~ sample40)
    auto dataset = loadPatternData(40);

    const int epochs = 5;           // 에폭 수 (예시)
    const int batchSize = 10;       // 미니배치 크기

    std::mt19937 rng(std::random_device{}());

    // 학습 루프
    for(int epoch=0; epoch<epochs; ++epoch){
        std::shuffle(dataset.begin(), dataset.end(), rng);

        for(size_t i=0; i<dataset.size(); i += batchSize){
            size_t end = std::min(i+batchSize, dataset.size());
            for(size_t j=i; j<end; ++j){
                auto& inputMat = dataset[j].first;
                auto& targetMat = dataset[j].second;

                // 순전파
                inputlayer.feedforward(inputMat);
                input.pushInput(inputlayer.getOutput());
                input.Active();

                hiddenlayer1.feedforward(input.getOutput());
                hidden1.pushInput(hiddenlayer1.getOutput());
                hidden1.Active();

                outputlayer.feedforward(hidden1.getOutput());
                output.pushInput(outputlayer.getOutput());
                output.Active();

                // 손실 계산 및 역전파
                loss.pushTarget(targetMat);
                loss.pushOutput(output.getOutput());
                d_matrix<double> grad = loss.getGrad();

                outputlayer.backprop(nullptr, grad, output.d_Active(outputlayer.getOutput()));
                // 다음 계층 정보를 포인터로 전달하면 내부에서 next->delta를 활용하여
                // 역전파가 진행된다. 외부에서 delta에 접근할 필요는 없다.
                d_matrix<double> dummy(1,1); // 사용되지 않음
                hiddenlayer1.backprop(&outputlayer, dummy, hidden1.d_Active(hiddenlayer1.getOutput()));
                inputlayer.backprop(&hiddenlayer1, dummy, input.d_Active(inputlayer.getOutput()));
            }
        }

        std::cout << "[Epoch " << epoch+1 << "] 학습 완료" << std::endl;
    }

    // 학습 후 각 샘플에 대한 예측 결과 출력
    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto& inputMat = dataset[idx].first;

        inputlayer.feedforward(inputMat);
        input.pushInput(inputlayer.getOutput());
        input.Active();
        hiddenlayer1.feedforward(input.getOutput());
        hidden1.pushInput(hiddenlayer1.getOutput());
        hidden1.Active();
        outputlayer.feedforward(hidden1.getOutput());
        output.pushInput(outputlayer.getOutput());
        output.Active();

        d_matrix<double> pred = output.getOutput();
        pred.cpyToHost();

        std::cout << "=== sample " << idx+1 << " 결과 ===" << std::endl;
        for(int r=0;r<BOARDHEIGHT;r++){
            for(int c=0;c<BOARDWIDTH;c++){
                std::cout << (pred(r*BOARDWIDTH+c,0) > 0.5 ? '1' : '0');
            }
            std::cout << '\n';
        }
        std::cout << std::endl;
    }

    return 0;
}