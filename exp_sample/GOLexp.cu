#include "hip/hip_runtime.h"
#include "database.hpp"
#include "perceptron.hpp"
#include <vector>
#include <fstream>
#include <string>
#include <algorithm>
#include <random>

// 입력 패턴 크기 (10x10)
const int WIDTH = 10;
const int HEIGHT = 10;

// MLP 구조 정의
Adam inputlayer(WIDTH * HEIGHT, 256, 0.001, InitType::He);
ActivateLayer input(256, 1, ActivationType::LReLU);
Adam hiddenlayer1(256, 256, 0.001, InitType::He);
ActivateLayer hidden1(256, 1, ActivationType::LReLU);
Adam outputlayer(256, BOARDHEIGHT * BOARDWIDTH, 0.001, InitType::He);
ActivateLayer output(BOARDHEIGHT * BOARDWIDTH, 1, ActivationType::LReLU);
LossLayer loss(BOARDHEIGHT * BOARDWIDTH, 1, LossType::MSE);

// 데이터 로딩
std::vector<std::pair<d_matrix<double>, d_matrix<double>>> loadPatternData() {
    std::vector<std::pair<d_matrix<double>, d_matrix<double>>> dataset;
    for (int i = 1; i <= 40; ++i) {
        std::ifstream fin("dataset/sample" + std::to_string(i) + ".txt");
        if (!fin) continue;

        d_matrix<double> in(WIDTH * HEIGHT, 1);
        std::string line;
        for (int r = 0; r < HEIGHT; ++r) {
            std::getline(fin, line);
            for (int c = 0; c < WIDTH; ++c)
                in(r * WIDTH + c, 0) = line[c] - '0';
        }

        std::getline(fin, line); // label (사용하지 않음)
        std::getline(fin, line); // 빈 줄

        d_matrix<double> out(BOARDHEIGHT * BOARDWIDTH, 1);
        for (int r = 0; r < BOARDHEIGHT; ++r) {
            std::getline(fin, line);
            for (int c = 0; c < BOARDWIDTH; ++c)
                out(r * BOARDWIDTH + c, 0) = line[c] - '0';
        }

        dataset.emplace_back(in, out);
    }
    return dataset;
}

// 순전파 후 결과 반환
d_matrix<double> forward(const d_matrix<double>& in) {
    inputlayer.feedforward(in);
    input.pushInput(inputlayer.getOutput());
    input.Active();

    hiddenlayer1.feedforward(input.getOutput());
    hidden1.pushInput(hiddenlayer1.getOutput());
    hidden1.Active();

    outputlayer.feedforward(hidden1.getOutput());
    output.pushInput(outputlayer.getOutput());
    output.Active();

    return output.getOutput();
}

// 학습 루프
void train(std::vector<std::pair<d_matrix<double>, d_matrix<double>>>& data) {
    const int batch = 10;
    const int epoch = 1;

    for (int e = 0; e < epoch; ++e) {
        std::shuffle(data.begin(), data.end(), std::mt19937{e});
        for (size_t i = 0; i < data.size(); i += batch) {
            size_t end = std::min(i + batch, data.size());
            for (size_t j = i; j < end; ++j) {
                auto& in = data[j].first;
                auto& target = data[j].second;

                d_matrix<double> pred = forward(in);

                loss.pushTarget(target);
                loss.pushOutput(pred);
                d_matrix<double> grad = loss.getGrad();

                outputlayer.backprop(nullptr, grad, output.d_Active(outputlayer.getOutput()));

                d_matrix<double> dummy(hiddenlayer1.getOutput().getRow(), 1);
                dummy.fill(0);
                hiddenlayer1.backprop(&outputlayer, dummy, hidden1.d_Active(hiddenlayer1.getOutput()));
                inputlayer.backprop(&hiddenlayer1, dummy, input.d_Active(inputlayer.getOutput()));
            }
        }
    }
}

// 예측 결과 출력
void printBoard(const d_matrix<double>& board) {
    for (int r = 0; r < BOARDHEIGHT; ++r) {
        for (int c = 0; c < BOARDWIDTH; ++c) {
            std::cout << (board(r * BOARDWIDTH + c, 0) > 0.5 ? '1' : '0');
        }
        std::cout << '\n';
    }
}

int main() {
    auto dataset = loadPatternData();
    train(dataset);

    if (!dataset.empty()) {
        d_matrix<double> result = forward(dataset[0].first);
        result.cpyToHost();
        printBoard(result);
    }

    return 0;
}