#include "hip/hip_runtime.h"
#include <database.hpp>
#include <perceptron.hpp>



const std::string path = "../dataset/result";

// 데이터셋 로드 함수: dataset/sampleN.txt 파일에서
// 10x10 입력 패턴과 300x300 결과 패턴을 읽어온다.
std::vector<std::pair<d_matrix<double>, d_matrix<double>>> loadPatternData(int count){
    std::vector<std::pair<d_matrix<double>, d_matrix<double>>> data;
    data.reserve(count);

    for(int idx = 1; idx <= count; ++idx){
        std::string path = "../dataset/sample" + std::to_string(idx) + ".txt";
        std::ifstream fin(path);
        if(!fin){
            std::cerr << "파일을 열 수 없습니다: " << path << std::endl;
            continue;
        }

        d_matrix<double> inputMat(100,1);      // 10x10 패턴
        d_matrix<double> outputMat(BOARDHEIGHT*BOARDWIDTH,1); // 300x300 결과 패턴

        std::string line;
        // 입력 패턴 읽기
        for(int r=0;r<10 && std::getline(fin,line); ++r){
            for(int c=0;c<10 && c<(int)line.size(); ++c){
                inputMat(r*10+c,0) = line[c]-'0';
            }
        }

        // 라벨(생존 칸 수)은 현재 사용하지 않으므로 읽기만 하고 무시
        std::getline(fin,line); // label
        std::getline(fin,line); // blank line

        // 결과 패턴 읽기 (300줄)
        for(int r=0;r<BOARDHEIGHT && std::getline(fin,line); ++r){
            for(int c=0;c<BOARDWIDTH && c<(int)line.size(); ++c){
                outputMat(r*BOARDWIDTH+c,0) = line[c]-'0';
            }
        }

        data.emplace_back(inputMat, outputMat);
    }

    return data;
}

int main(){
    Adam inputlayer(100, 256, 0.011, InitType::Xavier);
    ActivateLayer input(256, 1, ActivationType::Tanh);
    Adam hiddenlayer1(256, 512, 0.011, InitType::Xavier);
    ActivateLayer hidden1(512, 1, ActivationType::Tanh);
    Adam hiddenlayer2(512, 512, 0.011, InitType::Xavier);
    ActivateLayer hidden2(512, 1, ActivationType::Tanh);
    Adam outputlayer(512, BOARDHEIGHT*BOARDWIDTH, 0.011, InitType::Xavier);
    ActivateLayer output(BOARDHEIGHT*BOARDWIDTH, 1, ActivationType::Tanh);
    LossLayer loss(BOARDHEIGHT*BOARDWIDTH, 1, LossType::CrossEntropy);

    auto dataset = loadPatternData(SEMPLE);

    std::filesystem::create_directories(path);

    const char *commend1 = "find ../dataset/result -type f -delete";

    std::system(commend1);

    const int epochs = 20;           // 에폭 수 (예시)
    const int batchSize = 10;       // 미니배치 크기

    std::mt19937 rng(std::random_device{}());

    // 학습 루프
    for(int epoch=0; epoch<epochs; ++epoch){
        auto startTime = std::chrono::steady_clock::now();
        std::shuffle(dataset.begin(), dataset.end(), rng);

                double totalLoss = 0.0;
                size_t sampleCount = 0;

        for(size_t i=0; i<dataset.size(); i += batchSize){
            size_t end = std::min(i+batchSize, dataset.size());
            for(size_t j=i; j<end; ++j){
                auto& inputMat = dataset[j].first;
                auto& targetMat = dataset[j].second;

                // 순전파
                inputlayer.feedforward(inputMat);
                input.pushInput(inputlayer.getOutput());
                input.Active();

                hiddenlayer1.feedforward(input.getOutput());
                hidden1.pushInput(hiddenlayer1.getOutput());
                hidden1.Active();

                hiddenlayer2.feedforward(hiddenlayer1.getOutput());
                hidden2.pushInput(hiddenlayer2.getOutput());
                hidden2.Active();

                outputlayer.feedforward(hidden2.getOutput());
                output.pushInput(outputlayer.getOutput());
                output.Active();

                // 손실 계산 및 역전파
                loss.pushTarget(targetMat);
                loss.pushOutput(output.getOutput());

                totalLoss += loss.getLoss();
                ++sampleCount;

                outputlayer.backprop(nullptr, loss.getGrad(), output.d_Active(outputlayer.getOutput()));
                // 다음 계층 정보를 포인터로 전달하면 내부에서 next->delta를 활용하여
                // 역전파가 진행된다. 외부에서 delta에 접근할 필요는 없다.
                d_matrix<double> dummy(1,1); // 사용되지 않음
                hiddenlayer2.backprop(&outputlayer, dummy, hidden2.d_Active(hiddenlayer2.getOutput()));
                hiddenlayer1.backprop(&hiddenlayer2, dummy, hidden1.d_Active(hiddenlayer1.getOutput()));
                inputlayer.backprop(&hiddenlayer1, dummy, input.d_Active(inputlayer.getOutput()));

                printProgressBar(j, dataset.size(), startTime, "Epoch" + std::to_string(epoch+1) + "진행중..." + "(loss:" + std::to_string(loss.getLoss()) + ")");
            }
        }

        double avgLoss = totalLoss / static_cast<double>(sampleCount);

        std::cout << "✅ Epoch " << (epoch+1)
                  << " 완료! (소요 "
                  << std::chrono::duration_cast<std::chrono::seconds>(
                         std::chrono::steady_clock::now() - startTime
                     ).count()
                  << "초),"
                  << "loss:"
                  << avgLoss
                  << "                                                                                                                                          "
                  << std::endl;
    }

    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto& inputMat = dataset[idx].first;

        inputlayer.feedforward(inputMat);
        input.pushInput(inputlayer.getOutput());
        input.Active();
        hiddenlayer1.feedforward(input.getOutput());
        hidden1.pushInput(hiddenlayer1.getOutput());
        hidden1.Active();
        hiddenlayer2.feedforward(hiddenlayer1.getOutput());
        hidden2.pushInput(hiddenlayer2.getOutput());
        hidden2.Active();
        outputlayer.feedforward(hidden2.getOutput());
        output.pushInput(outputlayer.getOutput());
        output.Active();

        d_matrix<double> pred = output.getOutput();
        pred.cpyToHost();

        std::ofstream result(path + "/semple_result" + std::to_string(idx+1) + ".txt");

        result << "=== sample " << idx+1 << " 결과 ===" << std::endl;
        for(int r=0;r<BOARDHEIGHT;r++){
            for(int c=0;c<BOARDWIDTH;c++){
                result << (pred(r*BOARDWIDTH+c,0) > 0.1 ? '1' : '0');
            }
            result << '\n';
        }
        result << '\n';
        result.close();
    }

    return 0;
}