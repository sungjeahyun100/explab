
#include <hip/hip_runtime.h>
#include <filesystem>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <system_error>

int main() {
    const std::string inputDir  = "../graph/count_ver_loss";
    const std::string outputDir = "../graph/count_ver_loss_deriv";

    // 출력 디렉터리 생성
    try {
        std::filesystem::create_directories(outputDir);
    } catch (const std::filesystem::filesystem_error &e) {
        std::cerr << "Directory creation failed: " << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    // 입력 디렉터리 순회
    for (const auto &entry : std::filesystem::directory_iterator(inputDir)) {
        if (!entry.is_regular_file()) continue;
        auto inPath = entry.path();
        if (inPath.extension() != ".txt") continue;

        // 출력 파일명
        std::string outName = "deriv_" + inPath.filename().string();
        std::filesystem::path outPath = std::filesystem::path(outputDir) / outName;

        std::ifstream fin(inPath);
        std::ofstream fout(outPath);
        if (!fin.is_open() || !fout.is_open()) {
            std::cerr << "Failed to open input or output: "
                      << inPath << " or " << outPath << std::endl;
            continue;
        }

        std::string line;
        double epoch = 0.0, loss = 0.0;
        double prev_loss = 0.0;
        bool first = true;

        // 데이터 라인별 읽기, 주석(#) 및 빈 라인 무시
        while (std::getline(fin, line)) {
            if (line.empty() || line[0] == '#') continue;
            std::istringstream ss(line);
            if (!(ss >> epoch >> loss)) continue;

            double dloss = first ? 0.0 : (loss - prev_loss);
            fout << epoch << ' ' << dloss << '\n';

            prev_loss = loss;
            first = false;
        }

        fin.close();
        fout.close();

        // 파일 권한 설정: 읽기/쓰기 추가
        std::error_code ec;
        std::filesystem::permissions(outPath,
            std::filesystem::perms::owner_read  | std::filesystem::perms::owner_write  |
            std::filesystem::perms::group_read  | std::filesystem::perms::group_write  |
            std::filesystem::perms::others_read | std::filesystem::perms::others_write,
            std::filesystem::perm_options::add, ec);
        if (ec) {
            std::cerr << "Permission change failed for " << outName
                      << ": " << ec.message() << std::endl;
        }

        std::cout << "Processed: " << inPath.filename()
                  << " -> " << outName << std::endl;
    }

    return 0;
}
