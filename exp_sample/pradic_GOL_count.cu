#include "hip/hip_runtime.h"
#include <database.hpp>
#include <perceptron.hpp>
#include <filesystem>
#include <fstream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <cmath>

const std::string result_path = "../dataset/result";

int main(){
    auto dataset = LoadingData();

    std::filesystem::create_directories(result_path);
    const char *cmd = "find ../dataset/result -type f -delete";
    std::system(cmd);

    Adam layer1(100, 128, 0.01, InitType::Xavier);
    ActivateLayer act1(128, 1, ActivationType::Tanh);
    Adam layer2(128, 64, 0.01, InitType::Xavier);
    ActivateLayer act2(64, 1, ActivationType::Tanh);
    Adam outputLayer(64, 1, 0.01, InitType::Xavier);
    ActivateLayer outAct(1, 1, ActivationType::Identity);
    LossLayer loss(1, 1, LossType::MSE);

    const int epochs = 20;
    const int batchSize = 10;
    std::mt19937 rng(std::random_device{}());

    for(int epoch=0; epoch<epochs; ++epoch){
        auto startTime = std::chrono::steady_clock::now();
        std::shuffle(dataset.begin(), dataset.end(), rng);

        for(size_t i=0; i<dataset.size(); i+=batchSize){
            size_t end = std::min(i+batchSize, dataset.size());
            for(size_t j=i; j<end; ++j){
                auto &inputMat = dataset[j].first;
                auto &targetMat = dataset[j].second;

                layer1.feedforward(inputMat);
                act1.pushInput(layer1.getOutput());
                act1.Active();

                layer2.feedforward(act1.getOutput());
                act2.pushInput(layer2.getOutput());
                act2.Active();

                outputLayer.feedforward(act2.getOutput());
                outAct.pushInput(outputLayer.getOutput());
                outAct.Active();

                loss.pushTarget(targetMat);
                loss.pushOutput(outAct.getOutput());

                outputLayer.backprop(nullptr, loss.getGrad(), outAct.d_Active(outputLayer.getOutput()));
                d_matrix<double> dummy(1,1);
                layer2.backprop(&outputLayer, dummy, act2.d_Active(layer2.getOutput()));
                layer1.backprop(&layer2, dummy, act1.d_Active(layer1.getOutput()));

                printProgressBar(j, dataset.size(), startTime, "Epoch" + std::to_string(epoch+1) + " 진행중...(loss:" + std::to_string(loss.getLoss()) + ")");
            }
        }
        std::cout << "\n✅ Epoch " << (epoch+1) << " 완료" << std::endl;
    }

    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto &inputMat = dataset[idx].first;

        layer1.feedforward(inputMat);
        act1.pushInput(layer1.getOutput());
        act1.Active();
        layer2.feedforward(act1.getOutput());
        act2.pushInput(layer2.getOutput());
        act2.Active();
        outputLayer.feedforward(act2.getOutput());
        outAct.pushInput(outputLayer.getOutput());
        outAct.Active();

        d_matrix<double> pred = outAct.getOutput();
        pred.cpyToHost();
        int count = static_cast<int>(std::round(pred(0,0)));

        std::ofstream ofs(result_path + "/sample_count_ver_" + std::to_string(idx+1) + ".txt");
        ofs << "=== sample " << idx+1 << " 결과 ===\n";
        ofs << count << "\n";
        ofs.close();
    }

    return 0;
}
