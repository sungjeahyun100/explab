#include "hip/hip_runtime.h"
#include <database.hpp>
#include <perceptron.hpp>
#include <filesystem>
#include <fstream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <cmath>

const std::string result_path = "../dataset/result";

int main(){
    auto dataset = LoadingData();

    std::filesystem::create_directories(result_path);
    const char *cmd = "find ../dataset/result -type f -delete";
    std::system(cmd);

    Adam layer1(100, 128, 0.0001, InitType::He);
    ActivateLayer act1(128, 1, ActivationType::LReLU);
    Adam layer2(128, 64, 0.0001, InitType::He);
    ActivateLayer act2(64, 1, ActivationType::LReLU);
    Adam outputLayer(64, BIT_WIDTH, 0.0001, InitType::He);
    ActivateLayer outAct(BIT_WIDTH, 1, ActivationType::LReLU);
    LossLayer loss(BIT_WIDTH, 1, LossType::MSE);

    const int epochs = 1000;
    const int batchSize = 10;
    std::mt19937 rng(std::random_device{}());
    
    for(int epoch = 0; epoch < epochs; ++epoch){
        auto startTime = std::chrono::steady_clock::now();
    
        // 1) 에폭 시작 시 한 번만 shuffle
        std::shuffle(dataset.begin(), dataset.end(), rng);
    
        double totalLoss = 0.0;
        size_t sampleCount = 0;
    
        // 2) 배치별 학습
        for(size_t i = 0; i < dataset.size(); i += batchSize){
            size_t end = std::min(i + batchSize, dataset.size());
    
            for(size_t j = i; j < end; ++j){
                auto &inputMat  = dataset[j].first;
                auto &targetMat = dataset[j].second;
    
                // (b) 순전파
                layer1.feedforward(inputMat);
                act1.pushInput(layer1.getOutput()); act1.Active();
                layer2.feedforward(act1.getOutput());
                act2.pushInput(layer2.getOutput()); act2.Active();
                outputLayer.feedforward(act2.getOutput());
                outAct.pushInput(outputLayer.getOutput()); outAct.Active();
    
                // (c) 손실 계산
                loss.pushTarget(targetMat);
                loss.pushOutput(outAct.getOutput());
                double L = loss.getLoss();
                totalLoss += L;
                ++sampleCount;
    
                // (d) 역전파
                auto Grad = loss.getGrad();
                outputLayer.backprop(nullptr, Grad, outAct.d_Active(outputLayer.getOutput()));
                d_matrix<double> dummy(1,1);
                layer2.backprop(&outputLayer, dummy, act2.d_Active(layer2.getOutput()));
                layer1.backprop(&layer2, dummy, act1.d_Active(layer1.getOutput()));
    
                // (e) 진행 표시
                printProgressBar(j, dataset.size(), startTime, "Epoch " + std::to_string(epoch+1) + " 진행중...(loss:" + std::to_string(L) + ")");
            }
        }
    
        // 3) 에폭 단위 평균 손실 계산
        double avgLoss = totalLoss / static_cast<double>(sampleCount);
    
        std::cout << "✅ Epoch " << (epoch+1)
                  << " 완료! (소요 "
                  << std::chrono::duration_cast<std::chrono::seconds>(
                         std::chrono::steady_clock::now() - startTime
                     ).count()
                  << "초), 평균 손실: "
                  << avgLoss
                  << "                                                                                                                                          "
                  << std::endl;
    }


    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto &inputMat = dataset[idx].first;
    
        // 순전파
        layer1.feedforward(inputMat);
        act1.pushInput(layer1.getOutput()); 
        act1.Active();
        layer2.feedforward(act1.getOutput());
        act2.pushInput(layer2.getOutput()); 
        act2.Active();
        outputLayer.feedforward(act2.getOutput());
        outAct.pushInput(outputLayer.getOutput()); 
        outAct.Active();
    
        // 예측값 복사
        d_matrix<double> pred = outAct.getOutput();
        pred.cpyToHost();
    
        // 비트 예측 및 정수값 복원
        int count = 0;
        for(int b = 0; b < BIT_WIDTH; ++b){
            // sigmoid 출력이니 0.5 기준으로 0/1 결정
            int bit = (pred(b,0) > 0.5) ? 1 : 0;
            count |= (bit << b);  // 2^b 만큼 더하기
        }
    
        // 결과 저장
        std::ofstream ofs(result_path + "/sample_count_bin_" + std::to_string(idx+1) + ".txt");
        if(!ofs){
            std::cerr << "파일 열기 실패: sample " << (idx+1) << "\n";
            continue;
        }
        ofs << "=== sample " << (idx+1) << " 결과 ===\n";
        ofs << count << "\n";
        ofs.close();
    }
    
    return 0;
}
