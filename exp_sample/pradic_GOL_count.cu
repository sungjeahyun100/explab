#include "hip/hip_runtime.h"
#include <database.hpp>
#include <perceptron.hpp>
#include <filesystem>
#include <fstream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <cmath>

const std::string result_path = "../dataset/result";
const std::string graph_path = "../graph/count_ver_loss";

int main(){
    auto dataset = LoadingData();

    std::filesystem::create_directories(result_path);
    std::filesystem::create_directories(graph_path); 
    const char *cmd = "find ../dataset/result -type f -delete";
    std::system(cmd);

    std::ofstream loss_ofs(graph_path + "/loss_data_He_LReLU_MSE_batch50.txt");  // ← 추가
    loss_ofs << "# epoch loss\n"; 

    Adam inputlayer(100, 512, 0.0001, InitType::He);
    ActivateLayer inputAct(512, 1, ActivationType::LReLU);
    Adam hiddenlayer1(512, 512, 0.0001, InitType::He);
    ActivateLayer hiddenAct1(512, 1, ActivationType::LReLU);
    Adam hiddenlayer2(512, 128, 0.0001, InitType::He);
    ActivateLayer hiddenAct2(128, 1, ActivationType::LReLU);
    Adam outputLayer(128, BIT_WIDTH, 0.0001, InitType::He);
    ActivateLayer outAct(BIT_WIDTH, 1, ActivationType::LReLU);
    LossLayer loss(BIT_WIDTH, 1, LossType::MSE);

    const int epochs = 100;
    const int batchSize = 50;
    std::mt19937 rng(std::random_device{}());
    
    for(int epoch = 0; epoch < epochs; ++epoch){
        auto startTime = std::chrono::steady_clock::now();
    
        // 1) 에폭 시작 시 한 번만 shuffle
        std::shuffle(dataset.begin(), dataset.end(), rng);
    
        double totalLoss = 0.0;
        size_t sampleCount = 0;
    
        // 2) 배치별 학습
        for(size_t i = 0; i < dataset.size(); i += batchSize){
            size_t end = std::min(i + batchSize, dataset.size());
    
            for(size_t j = i; j < end; ++j){
                auto &inputMat  = dataset[j].first;
                auto &targetMat = dataset[j].second;
    
                // (b) 순전파
                inputlayer.feedforward(inputMat);
                inputAct.pushInput(inputlayer.getOutput()); inputAct.Active();
                hiddenlayer1.feedforward(inputAct.getOutput());
                hiddenAct1.pushInput(hiddenlayer1.getOutput()); hiddenAct1.Active();
                hiddenlayer2.feedforward(hiddenAct1.getOutput());
                hiddenAct2.pushInput(hiddenlayer2.getOutput()); hiddenAct2.Active();
                outputLayer.feedforward(hiddenAct2.getOutput());
                outAct.pushInput(outputLayer.getOutput()); outAct.Active();
    
                // (c) 손실 계산
                loss.pushTarget(targetMat);
                loss.pushOutput(outAct.getOutput());
                double L = loss.getLoss();
                totalLoss += L;
                ++sampleCount;
    
                // (d) 역전파
                auto Grad = loss.getGrad();
                outputLayer.backprop(nullptr, Grad, outAct.d_Active(outputLayer.getOutput()));
                d_matrix<double> dummy(1,1);
                hiddenlayer2.backprop(&outputLayer, dummy, hiddenAct2.d_Active(hiddenlayer2.getOutput()));
                hiddenlayer1.backprop(&hiddenlayer2, dummy, hiddenAct1.d_Active(hiddenlayer1.getOutput()));
                inputlayer.backprop(&hiddenlayer1, dummy, inputAct.d_Active(inputlayer.getOutput()));
    
                // (e) 진행 표시
                printProgressBar(j, dataset.size(), startTime, "Epoch " + std::to_string(epoch+1) + " 진행중...(loss:" + std::to_string(totalLoss/1000) + ")");
            }
        }
    
        // 3) 에폭 단위 평균 손실 계산
        double avgLoss = totalLoss / static_cast<double>(sampleCount);

        loss_ofs << (epoch+1) << " " << avgLoss << "\n";
    
        std::cout << "✅ Epoch " << (epoch+1)
                  << " 완료! (소요 "
                  << std::chrono::duration_cast<std::chrono::seconds>(
                         std::chrono::steady_clock::now() - startTime
                     ).count()
                  << "초), 평균 손실: "
                  << avgLoss
                  << "                                                                                                                                          "
                  << std::endl;
    }

    loss_ofs.close();


    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto &inputMat = dataset[idx].first;
    
        // 순전파
        inputlayer.feedforward(inputMat);
        inputAct.pushInput(inputlayer.getOutput()); inputAct.Active();
        hiddenlayer1.feedforward(inputAct.getOutput());
        hiddenAct1.pushInput(hiddenlayer1.getOutput()); hiddenAct1.Active();
        hiddenlayer2.feedforward(hiddenAct1.getOutput());
        hiddenAct2.pushInput(hiddenlayer2.getOutput()); hiddenAct2.Active();
        outputLayer.feedforward(hiddenAct2.getOutput());
        outAct.pushInput(outputLayer.getOutput()); outAct.Active();
    
        // 예측값 복사
        d_matrix<double> pred = outAct.getOutput();
        pred.cpyToHost();
    
        // 비트 예측 및 정수값 복원
        int count = 0;
        for(int b = 0; b < BIT_WIDTH; ++b){
            // sigmoid 출력이니 0.5 기준으로 0/1 결정
            int bit = (pred(b,0) > 0.5) ? 1 : 0;
            count |= (bit << b);  // 2^b 만큼 더하기
        }
    
        // 결과 저장
        std::ofstream ofs(result_path + "/sample_count_bin_" + std::to_string(idx+1) + ".txt");
        if(!ofs){
            std::cerr << "파일 열기 실패: sample " << (idx+1) << "\n";
            continue;
        }
        ofs << "=== sample " << (idx+1) << " 결과 ===\n";
        ofs << count << "\n";
        ofs << pred(0, 0) << "," << pred(1, 0) << "," << pred(2, 0) << ","<< pred(3, 0) << "," << pred(4, 0) << "," << pred(5, 0) << "," << pred(6, 0) << "," << pred(7, 0) << "," << "\n";
        ofs.close();
    }
    
    return 0;
}
