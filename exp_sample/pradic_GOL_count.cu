#include "hip/hip_runtime.h"
#include <database.hpp>
#include <perceptron.hpp>
#include <filesystem>
#include <fstream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <cmath>

const std::string result_path = "../dataset/result";

int main(){
    auto dataset = LoadingData();

    std::filesystem::create_directories(result_path);
    const char *cmd = "find ../dataset/result -type f -delete";
    std::system(cmd);

    Adam layer1(100, 128, 0.0005, InitType::He);
    ActivateLayer act1(128, 1, ActivationType::LReLU);
    Adam layer2(128, 64, 0.0005, InitType::He);
    ActivateLayer act2(64, 1, ActivationType::LReLU);
    Adam outputLayer(64, BIT_WIDTH, 0.0005, InitType::He);
    ActivateLayer outAct(BIT_WIDTH, 1, ActivationType::LReLU);
    LossLayer loss(BIT_WIDTH, 1, LossType::MSE);

    const int epochs = 400;
    const int batchSize = 50;

    for(int epoch=0; epoch<epochs; ++epoch){
        auto startTime = std::chrono::steady_clock::now();

        for(size_t i=0; i<dataset.size(); i+=batchSize){
            size_t end = std::min(i+batchSize, dataset.size());
            for(size_t j=i; j<end; ++j){
                auto &inputMat = dataset[j].first;
                auto &targetMat = dataset[j].second;

                layer1.feedforward(inputMat);
                act1.pushInput(layer1.getOutput());
                act1.Active();

                layer2.feedforward(act1.getOutput());
                act2.pushInput(layer2.getOutput());
                act2.Active();

                outputLayer.feedforward(act2.getOutput());
                outAct.pushInput(outputLayer.getOutput());
                outAct.Active();

                loss.pushTarget(targetMat);
                loss.pushOutput(outAct.getOutput());

                auto Grad = loss.getGrad();
                auto Loss = loss.getLoss();

                outputLayer.backprop(nullptr, Grad, outAct.d_Active(outputLayer.getOutput()));
                d_matrix<double> dummy(1,1);
                layer2.backprop(&outputLayer, dummy, act2.d_Active(layer2.getOutput()));
                layer1.backprop(&layer2, dummy, act1.d_Active(layer1.getOutput()));

                printProgressBar(j, dataset.size(), startTime, "Epoch" + std::to_string(epoch+1) + " 진행중...(loss:" + std::to_string(Loss) + ")");
            }
        }
        std::cout << "✅ Epoch " << (epoch+1)
                  << " 완료! (소요 "
                  << std::chrono::duration_cast<std::chrono::seconds>(
                         std::chrono::steady_clock::now() - startTime
                     ).count()
                  << "초),"
                  << "loss:"
                  << loss.getLoss()
                  << "                                                                                                                                          "
                  << std::endl;
    }

    for(size_t idx=0; idx<dataset.size(); ++idx){
        auto &inputMat = dataset[idx].first;
    
        // 순전파
        layer1.feedforward(inputMat);
        act1.pushInput(layer1.getOutput()); 
        act1.Active();
        layer2.feedforward(act1.getOutput());
        act2.pushInput(layer2.getOutput()); 
        act2.Active();
        outputLayer.feedforward(act2.getOutput());
        outAct.pushInput(outputLayer.getOutput()); 
        outAct.Active();
    
        // 예측값 복사
        d_matrix<double> pred = outAct.getOutput();
        pred.cpyToHost();
    
        // 비트 예측 및 정수값 복원
        int count = 0;
        for(int b = 0; b < BIT_WIDTH; ++b){
            // sigmoid 출력이니 0.5 기준으로 0/1 결정
            int bit = (pred(b,0) > 0.5) ? 1 : 0;
            count |= (bit << b);  // 2^b 만큼 더하기
        }
    
        // 결과 저장
        std::ofstream ofs(result_path + "/sample_count_bin_" + std::to_string(idx+1) + ".txt");
        if(!ofs){
            std::cerr << "파일 열기 실패: sample " << (idx+1) << "\n";
            continue;
        }
        ofs << "=== sample " << (idx+1) << " 결과 ===\n";
        ofs << count << "\n";
        ofs.close();
    }
    
    return 0;
}
